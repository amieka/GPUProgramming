#include "hip/hip_runtime.h"

#include "Image.h"
#include <hip/hip_runtime.h>

//define the kernel function here
//kernel function for converting rgba to gray scale image
__global__
void rgba_to_greyscale(const u_char* const input,
                       u_char* output,
                       int numRows, int numCols)
{
    
   int col = blockIdx.y * blockDim.y + threadIdx.y;
   int row = blockIdx.x * blockDim.x + threadIdx.x;
   if ( row >= numRows || col >= numCols )
   {
      return;
   }
   	int offset_r = (row * numCols + col) * 3 + 1;
	int offset_g = (row * numCols + col) * 3 + 2;
	int offset_b = (row * numCols + col) * 3 + 3;
	float channelSum = .299f * input[offset_r] + .587f * input[offset_g] + .114f * input[offset_b];
	output[row * numCols + col] = channelSum;
}

void my_gray_scale(const u_char *input, u_char *output, int numRows, int numCols) {
	int row = 0, col = 0;
	for(row = 0; row < numRows; row++) {
		for(col = 0; col < numCols; col++) {
			int offset_r = (row * numCols + col) * 3 + 1;
			int offset_g = (row * numCols + col) * 3 + 2;
			int offset_b = (row * numCols + col) * 3 + 3;
			float channelSum = .299f * input[offset_r] + .587f * input[offset_g] + .114f * input[offset_b];
			output[row * numCols + col] = channelSum;
		}
	}
}

int main() {
	PPM *ppmImage = read_image("GreatWave.ppm");
	PPM *outImage = new PPM();
	int ppm_bytes = 0, pgm_bytes = 0;
	int numRows, numCols;
	u_char *d_gray = NULL;
	u_char *h_rgba = NULL;
	u_char *h_gray = NULL;
	u_char *rgba = NULL;
	if(!ppmImage) {
		printf("there was an error reading the image");
	}
	numRows = ppmImage->height;
	numCols = ppmImage->width;
	outImage->height = ppmImage->height;
	outImage->width = ppmImage->width;
	
	//printf("image data %d %d ", numRows, numCols);
	
	ppm_bytes = numCols * numRows * sizeof(u_char) * 3;
	pgm_bytes = numCols * numRows * sizeof(u_char);
	
	//allocate some memory to host data structures
	h_rgba = ppmImage->data;
	h_gray = (u_char *)malloc(pgm_bytes); // allocate some memory to the output image
	
	//allocate memory to device data structures
	hipMalloc((void**)&d_gray, pgm_bytes);
	hipMalloc((void**)&rgba, ppm_bytes);
	
	
	//memcpy stuff from host to device array 
	hipMemcpy(rgba, h_rgba, ppm_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_gray, h_gray, ppm_bytes, hipMemcpyHostToDevice);
	
	
	//set the block and grid size
	
	const dim3 blockSize(1,1);  //set a block size of 1
	const dim3 gridSize(numRows , numCols );  	
	
	//run the kernel function for converting rgba into gray scale image
	
	rgba_to_greyscale<<<gridSize, blockSize>>>(rgba, d_gray, numRows, numCols);
	//my_gray_scale(rgba, d_gray, numRows, numCols);
	
	//copy back data from device to host again
	
	hipMemcpy(h_gray, d_gray, pgm_bytes, hipMemcpyDeviceToHost);
	
	outImage->data = h_gray;
	//write out to a pgm file here
	printf("saving the pgm image here \n");
	save_pgm_image(outImage, "out.pgm");
	
	//printf("height %d\n", ppmImage->height);
	return 0;
}